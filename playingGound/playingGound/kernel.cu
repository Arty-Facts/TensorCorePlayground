#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <chrono>


#define BLOCKSIZE 32

template<typename T>
__global__ void matrixMultiplicationKernel(T* C, T* A, T* B,  unsigned int N) {

    unsigned int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int COL = blockIdx.x * blockDim.x + threadIdx.x;

    T tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (unsigned int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    //printf("tmp sum: %f\n", tmpSum);
    C[ROW * N + COL] = tmpSum;
    
}

template<typename T>
class matrixMultiplication
{
public:
    matrixMultiplication() = default;
    matrixMultiplication(T* c, const T* a, const T* b,  unsigned int n)
        : c{ c }, a{ a }, b{ b }, size{ n*n }, n{ n } {};

    void lanch()
    {
        dim3 threadsPerBlock(n, n);
        dim3 blocksPerGrid(1, 1);
        if (n > BLOCKSIZE) {
            threadsPerBlock.x = BLOCKSIZE;
            threadsPerBlock.y = BLOCKSIZE;
            blocksPerGrid.x = ceil(double(n) / double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(n) / double(threadsPerBlock.y));
        }
        matrixMultiplicationKernel<T> <<<blocksPerGrid, threadsPerBlock >>> (dev_c, dev_a, dev_b, n);
    }

    hipError_t setup() 
    {
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            return cudaStatus;
        }

        // Allocate GPU buffers for three vectors (two input, one output)    .
        cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(T));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(T));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(T));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(dev_a, a, size * sizeof(T), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }

        cudaStatus = hipMemcpy(dev_b, b, size * sizeof(T), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }
    }
    hipError_t therdown()
    {
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "matrixMultiplicationkernal launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching matrixMultiplicationkernal!\n", cudaStatus);
            return cudaStatus;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(c, dev_c, size * sizeof(T), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }
        return cudaStatus;
    }
    ~matrixMultiplication()
    {
        hipFree(dev_c);
        hipFree(dev_a);
        hipFree(dev_b);
    }

private:
    T* c;
    const T* a;
    const T* b;
    unsigned int size;
    unsigned int n;
    T* dev_a = 0;
    T* dev_b = 0;
    T* dev_c = 0;
    hipError_t cudaStatus;
};

int main()
{
    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    const int N = 128;
    const int SIZE = N * N;

    // Allocate memory on the host
    float a[SIZE];
    float b[SIZE];
    float c[SIZE];

    // Initialize matrices on the host
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i * N + j] = sin(i);
            b[i * N + j] = cos(j);
        }
    }

    matrixMultiplication<float> runner(c, a, b, N);
    printf("Start...\n");
    printf("Computing %d x %d matrix multiply...\n", N ,N);
    runner.setup();
    hipEvent_t myEventStart;
    hipEventCreate(&myEventStart);
    hipEventRecord(myEventStart, 0);
    hipEventSynchronize(myEventStart);

    runner.lanch();

    hipDeviceSynchronize();
    hipEvent_t myEventStop;
    hipEventCreate(&myEventStop);
    hipEventRecord(myEventStop, 0);
    hipEventSynchronize(myEventStop);
    float gpu_time;
    hipEventElapsedTime(&gpu_time, myEventStart, myEventStop);
    printf("GPU Computation time: %f\n", gpu_time);

    runner.therdown();

    float* cpu_C;
    cpu_C = new float[SIZE];
    float sum;

    auto start_cpu_time = std::chrono::high_resolution_clock::now();
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < N; col++) {
            sum = 0.f;
            for (int n = 0; n < N; n++) {
                sum += a[row * N + n] * b[n * N + col];
            }
            cpu_C[row * N + col] = sum;
        }
    }
    auto end_cpu_time = std::chrono::high_resolution_clock::now();
    float cpu_time = std::chrono::duration<double, std::milli>(end_cpu_time - start_cpu_time).count();
    printf("CPU Computation time: %f\n", cpu_time);
    double err = 0;
    // Check the result and make sure it is correct
    for (int ROW = 0; ROW < N; ROW++) {
        for (int COL = 0; COL < N; COL++) {
            // printf("cpu: %f -> gpu: %f\n", cpu_C[ROW * N + COL], c[ROW * N + COL]);
            err += cpu_C[ROW * N + COL] - c[ROW * N + COL];
        }
    }
    printf("Error:  %lf\n", err);
    delete cpu_C;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


