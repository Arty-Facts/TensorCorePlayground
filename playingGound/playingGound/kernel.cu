#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <chrono>


#define BLOCKSIZE 32
template<typename T>
__global__ void matrixMultiplicationKernel(T* C, T* A, T* B,  unsigned int N) {

    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    T tmpSum = 0;

    if (y < N && x < N) {
        // each thread computes one element of the block sub-matrix
        for (unsigned int i = 0; i < N; i++) {
            tmpSum += A[y * N + i] * B[i * N + x];
        }
    }
    //printf("tmp sum: %f\n", tmpSum);
    C[y * N + x] = tmpSum;
    
}
template<typename T>
__global__ void matmul_kernel(T* C, T* A, T* B, unsigned int N) {

    __shared__ T sA[BLOCKSIZE][BLOCKSIZE];
    __shared__ T sB[BLOCKSIZE][BLOCKSIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    T tmpSum = 0;
    int k, kb;

    for (k = 0; k < N; k += blockDim.x) {
        __syncthreads();
        sA[ty][tx] = A[y * N + k + tx];
        sB[ty][tx] = B[(k + ty) * N + x];
        __syncthreads();

        for (kb = 0; kb < blockDim.x; kb++) {
            tmpSum += sA[ty][kb] * sB[kb][tx];
        }

    }

    C[y * N + x] = tmpSum;
}

template<typename T>
class matrixMultiplication
{
public:
    matrixMultiplication() = default;
    matrixMultiplication(T* c, const T* a, const T* b,  unsigned int n)
        : c{ c }, a{ a }, b{ b }, size{ n*n }, n{ n } {};

    float lanch(void (*kernal)(T* C, T* A, T* B, unsigned int N) )
    {

        hipEvent_t myEventStart;
        hipEvent_t myEventStop;
        float gpu_time;

        dim3 threadsPerBlock(n, n);
        dim3 blocksPerGrid(1, 1);
        if (n > BLOCKSIZE) {
            threadsPerBlock.x = BLOCKSIZE;
            threadsPerBlock.y = BLOCKSIZE;
            blocksPerGrid.x = ceil(double(n) / double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(n) / double(threadsPerBlock.y));
        }
        // Time the run
        hipEventCreate(&myEventStart);
        hipEventRecord(myEventStart, 0);
        hipEventSynchronize(myEventStart);
        (*kernal) <<<blocksPerGrid, threadsPerBlock >>> (dev_c, dev_a, dev_b, n);
        // wait to be done
        hipDeviceSynchronize();
        hipEventCreate(&myEventStop);
        hipEventRecord(myEventStop, 0);
        hipEventSynchronize(myEventStop);
        hipEventElapsedTime(&gpu_time, myEventStart, myEventStop);
        return gpu_time;
    }

    hipError_t setup() 
    {
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            return cudaStatus;
        }

        // Allocate GPU buffers for three vectors (two input, one output)    .
        cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(T));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(T));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(T));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            return cudaStatus;
        }

        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(dev_a, a, size * sizeof(T), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }

        cudaStatus = hipMemcpy(dev_b, b, size * sizeof(T), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }
        return cudaStatus;
    }
    hipError_t therdown()
    {
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "kernal launch failed: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernal!\n", cudaStatus);
            return cudaStatus;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(c, dev_c, size * sizeof(T), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return cudaStatus;
        }
        return cudaStatus;
    }
    ~matrixMultiplication()
    {
        hipFree(dev_c);
        hipFree(dev_a);
        hipFree(dev_b);
    }

private:
    T* c;
    const T* a;
    const T* b;
    unsigned int size;
    unsigned int n;
    T* dev_a = 0;
    T* dev_b = 0;
    T* dev_c = 0;
    hipError_t cudaStatus;
};

template<typename T>
float matrixMultiplicationCPU(T* c, const T* a, const T* b, unsigned int N) {
    T sum;
    auto start_cpu_time = std::chrono::high_resolution_clock::now();
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < N; col++) {
            sum = 0.f;
            for (int n = 0; n < N; n++) {
                sum += a[row * N + n] * b[n * N + col];
            }
            c[row * N + col] = sum;
        }
    }
    auto end_cpu_time = std::chrono::high_resolution_clock::now();
    float cpu_time = std::chrono::duration<double, std::milli>(end_cpu_time - start_cpu_time).count();
    return cpu_time;

}
template<typename T>
double validate(T* val_c, T* c, unsigned int N) {
    double err = 0;
    // Check the result and make sure it is correct
    for (int ROW = 0; ROW < N; ROW++) {
        for (int COL = 0; COL < N; COL++) {
            //printf("cpu: %f -> gpu: %f\n", val_c[ROW * N + COL], c[ROW * N + COL]);
            err += val_c[ROW * N + COL] - c[ROW * N + COL];
        }
    }
    return err;
}

int main()
{
    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    const int N = 64;
    const int SIZE = N * N;

    // Allocate memory on the host
    float gpu_time;
    float cpu_time;
    double err;
    float a[SIZE];
    float b[SIZE];
    float c[SIZE];

    printf("Start...\n");
    printf("Computing %d x %d matrix ...\n", N ,N);
    // Initialize matrices on the host
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i * N + j] = sin(i);
            b[i * N + j] = cos(j);
        }
    }
    float* cpu_C;
    cpu_C = new float[SIZE];

    
    cpu_time = matrixMultiplicationCPU<float>(cpu_C, a, b, N);
    
    printf("CPU Computation time: %f\n", cpu_time);

    matrixMultiplication<float> runner(c, a, b, N);
    runner.setup();

    gpu_time = runner.lanch(&matrixMultiplicationKernel<float>);

    printf("matrixMultiplicationKernel GPU Computation time: %f\n", gpu_time);

    runner.therdown();

    // Check the result and make sure it is correct
    err = validate(c, cpu_C, N);
    printf("Error:  %lf\n", err);

    gpu_time =  runner.lanch(&matmul_kernel<float>);

    printf("matmul_kernel GPU Computation time: %f\n", gpu_time);

    runner.therdown();
    // Check the result and make sure it is correct
    err = validate(c, cpu_C, N);
    printf("Error:  %lf\n", err);
    delete cpu_C;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


